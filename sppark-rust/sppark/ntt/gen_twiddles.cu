#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

__global__
void generate_partial_twiddles(fr_t (*roots)[WINDOW_SIZE],
                               const fr_t root_of_unity)
{
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    assert(tid < WINDOW_SIZE);
    fr_t root;

    root = root_of_unity^tid;

    roots[0][tid] = root;

    for (int off = 1; off < WINDOW_NUM; off++) {
        for (int i = 0; i < LG_WINDOW_SIZE; i++)
#if defined(__CUDA_ARCH__)
            root.sqr();
#else
            root *= root;
#endif
        roots[off][tid] = root;
    }
}

__global__
void generate_all_twiddles(fr_t* d_radixX_twiddles, const fr_t root6,
                                                    const fr_t root7,
                                                    const fr_t root8,
                                                    const fr_t root9,
                                                    const fr_t root10)
{
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int pow;
    fr_t root_of_unity;

    if (tid < 64) {
        pow = tid;
        root_of_unity = root7;
    } else if (tid < 64 + 128) {
        pow = tid - 64;
        root_of_unity = root8;
    } else if (tid < 64 + 128 + 256) {
        pow = tid - 64 - 128;
        root_of_unity = root9;
    } else if (tid < 64 + 128 + 256 + 512) {
        pow = tid - 64 - 128 - 256;
        root_of_unity = root10;
    } else if (tid < 64 + 128 + 256 + 512 + 32) {
        pow = tid - 64 - 128 - 256 - 512;
        root_of_unity = root6;
    } else {
        assert(false);
    }

    d_radixX_twiddles[tid] = root_of_unity^pow;
}

#if !defined(FEATURE_BABY_BEAR) && !defined(FEATURE_GOLDILOCKS)
__launch_bounds__(512) __global__
void generate_radixX_twiddles_X(fr_t* d_radixX_twiddles_X, int n,
                                const fr_t root_of_unity)
{
    if (gridDim.x == 1) {
        d_radixX_twiddles_X[threadIdx.x] = fr_t::one();
        d_radixX_twiddles_X += blockDim.x;

        fr_t root0 = root_of_unity^threadIdx.x;

        d_radixX_twiddles_X[threadIdx.x] = root0;
        d_radixX_twiddles_X += blockDim.x;

        fr_t root1 = root0;

        for (int i = 2; i < n; i++) {
            root1 *= root0;
            d_radixX_twiddles_X[threadIdx.x] = root1;
            d_radixX_twiddles_X += blockDim.x;
        }
    } else {
        fr_t root0 = root_of_unity^(threadIdx.x * gridDim.x);

        unsigned int pow = blockIdx.x * threadIdx.x;
        unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

        fr_t root1 = root_of_unity^pow;

        d_radixX_twiddles_X[tid] = root1;
        d_radixX_twiddles_X += gridDim.x * blockDim.x;

        for (int i = gridDim.x; i < n; i += gridDim.x) {
            root1 *= root0;
            d_radixX_twiddles_X[tid] = root1;
            d_radixX_twiddles_X += gridDim.x * blockDim.x;
        }
    }
}
#endif
